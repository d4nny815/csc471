#include "hip/hip_runtime.h"
#include "camera.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cuda_help.h"

/**
 * @brief Constructs a Camera object with specified parameters.
 * @param aspect_ratio The aspect ratio of the image (width/height).
 * @param image_width The width of the output image in pixels.
 * @param samples_per_pixel The number of samples per pixel for anti-aliasing.
 * @param child_rays The number of child rays for each primary ray.
 * @param fov_deg The field of view in degrees, smaller zooms in
 * @param position The position of the camera in world space.
 * @param look_at The point the camera is looking at.
 * @param up_vector The up direction for the camera.
*/
Camera::Camera(char* filename, float aspect_ratio, size_t image_width, size_t samples_per_pixel, 
    size_t child_rays, float vfov_deg, float defocus_angle_deg, float focus_dist,
    const point3& position, const point3& look_at, const point3& up_vector) : 
    image_name(filename), image_width(image_width), 
    image_height(static_cast<size_t>(image_width / aspect_ratio)),
    aspect_ratio(aspect_ratio), samples_per_pixel(samples_per_pixel),
    scale_per_pixel(1.0f / samples_per_pixel), child_rays(child_rays), 
    defocus_angle_deg(defocus_angle_deg), focus_dist(focus_dist),
    pos(position), look_at(look_at), up_vector(up_vector) {

    // calc focal length
    // focal_length = (position - look_at).length();

    // calc viewport dimensions based on field of view
    vfov = degrees_to_radians(vfov_deg);
    float h = tan(vfov / 2);
    viewport_height = 2 * h * focus_dist;
    viewport_width = viewport_height * aspect_ratio;

    // calc orthonormal basis vectors for the camera coordinate system
    w = unit_vector(position - look_at);
    u = unit_vector(cross(up_vector, w));
    v = cross(w, u);

    // calc viewport edge vectors
    v_u = viewport_width * u;
    v_v = viewport_height * -v;

    // calc pixel delta vectors
    pixel_du = v_u / image_width;
    pixel_dv = v_v / image_height;

    // calc upper-left corner of the viewport
    viewport_upper_left = position - (focus_dist * w) - v_u / 2 - v_v / 2;
    pixel00_loc = viewport_upper_left + 0.5f * (pixel_du + pixel_dv);
    
    // Calculate the camera defocus disk basis vectors.
    auto defocus_radius = focus_dist * tan(degrees_to_radians(defocus_angle_deg / 2));
    defocus_disk_u = u * defocus_radius;
    defocus_disk_v = v * defocus_radius;

    frame_buffer = static_cast<uint32_t*>(malloc(sizeof(uint32_t) * 
        image_height * image_width));
}

/**
 * @brief shoot ray from cam to a pixel in the viewport
 * @param col current col in the viewport
 * @param row current row in the viewport
 * @return ray originating from the camera to the sampled pixel
 */
__host__ __device__ Ray Camera::get_ray(int col, int row) {
    // add random jitter to enable anti-aliasing
    // vec3 offset = sample_square(); 
    vec3 offset(0, 0, 0); 

    auto u = col + offset.x();
    auto v = row + offset.y();

    // get pixel location in the viewport
    vec3 pixel_sample = pixel00_loc + (u * pixel_du) + (v * pixel_dv);
    
    // dir from the cam to the sample pixel
    point3 ray_ori = (defocus_angle_deg <= 0) ? pos : defocus_disk_sample();
    vec3 ray_dir = pixel_sample - ray_ori;

    // printf("%d x %d %f x %f rOrigin <%f, %f, %f> rDir <%f, %f, %f>\n", 
    //     col, row, u, v,
    //     ray_ori.x(), ray_ori.y(), ray_ori.z(),
    //     ray_dir.x(), ray_dir.y(), ray_dir.z());
    
    return Ray(ray_ori, ray_dir);
}

/**
 * 
 */
__host__ __device__ point3 Camera::defocus_disk_sample() const {
    vec3 p = random_in_unit_disk();
    return pos + (p.x() * defocus_disk_u) + (p.y() * defocus_disk_v);
}

__host__ __device__ Color Camera::ray_color(const Ray& r, const size_t depth, const HittableList& world) 
    const {
    
    const Interval ray_interval(0.001, MY_INFINITY);
    Color accumulated_color(1.0, 1.0, 1.0); // Start with white (no attenuation)
    Ray current_ray = r;
    size_t current_depth = depth;

    while (current_depth > 0) {
        HitRecord hr;
        
        if (world.hit(current_ray, ray_interval, hr)) {
            Ray scattered;
            Color attenuation;
            
            if (hr.mat->scatter(current_ray, hr, attenuation, scattered)) {
                accumulated_color *= attenuation;  // Apply attenuation
                current_ray = scattered;  // Continue tracing the scattered ray
            } else {
                return BLACK * accumulated_color; // Absorb light (return black)
            }
        } else {
            // If the ray did not hit anything, return the sky color
            const Color SKY_BLUE = Color(0.5, 0.7, 1.0);
            vec3 unit_direction = unit_vector(current_ray.dir);
            auto a = 0.5 * (unit_direction.y() + 1.0);
            return accumulated_color * ((1.0 - a) * WHITE + a * SKY_BLUE);
        }

        current_depth--;
    }

    return Color(0, 0, 0); // Return black if maximum depth is reached
}

/**
 * @brief return vector with x = [-.5, .5] and y = [-.5, .5] 
 */
__host__ __device__ inline vec3 Camera::sample_square() {
    return vec3(rand_float() - 0.5, rand_float() - 0.5, 0);
}



__global__ void render_kernel(uint32_t* framebuffer, int image_width, int image_height, 
    int samples_per_pixel, Camera camera, HittableList world, 
    float scale_per_pixel) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= image_width || row >= image_height) return;

    Color pixel_color = Color(1, 0, 0);

    // // Monte Carlo sampling for anti-aliasing
    // for (int i = 0; i < samples_per_pixel; i++) {
    Ray r = camera.get_ray(col, row);
    pixel_color = camera.ray_color(r, camera.child_rays, world);
    // printf("%d x %d rOrigin <%f, %f, %f> rDir <%f, %f, %f> Color: [%f, %f, %f]\n", 
    //     col, row, 
    //     r.origin.x(), r.origin.y(), r.origin.z(),
    //     r.dir.x(), r.dir.y(), r.dir.z(),
    //     pixel_color.x(), pixel_color.y(), pixel_color.z());
    // }

    // Average the accumulated color
    // framebuffer[row * image_width + col] = pixel_color * scale_per_pixel;
    framebuffer[row * image_width + col] = write_color(pixel_color);
}

void Camera::render(const HittableList& world) {
    int num_pixels = image_width * image_height;
    uint32_t* d_framebuffer;
    
    // Allocate memory on the GPU
    CHECK_CUDA(hipMalloc(&d_framebuffer, num_pixels * sizeof(uint32_t)));

    // Define CUDA block and grid dimensions
    dim3 block_size(16, 16);
    dim3 grid_size((image_width + block_size.x - 1) / block_size.x, 
                   (image_height + block_size.y - 1) / block_size.y);

    // Launch CUDA kernel
    render_kernel<<<grid_size, block_size>>>(d_framebuffer, image_width, image_height, 
                                             samples_per_pixel, *this, world, scale_per_pixel);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy framebuffer from GPU to CPU
    CHECK_CUDA(hipMemcpy(frame_buffer, d_framebuffer, 
                num_pixels * sizeof(uint32_t), hipMemcpyDeviceToHost));

    // Write framebuffer to file or screen
    write_framebuffer();

    // Cleanup
    CHECK_CUDA(hipFree(d_framebuffer));
}

/**
 * @brief write the computed color value to the output in PPM format
 * 
 * takes a color vector, applies gamma correction, clamps
 * the values to the valid range, and converts them to 8bit color values and 
 * prints them for PPM format
 * 
 * @param k The color vector containing RGB values in linear space.
 */
__host__ __device__ uint32_t write_color(Color k) {
    float r = linear_to_gamma(k.x());
    float g = linear_to_gamma(k.y());
    float b = linear_to_gamma(k.z());

    uint8_t red = uint8_t(256 * Interval(0, .999).clamp(r));
    uint8_t green = uint8_t(256 * Interval(0, .999).clamp(g));
    uint8_t blue = uint8_t(256 * Interval(0, .999).clamp(b));

    uint32_t val = (red << 16) | (green << 8) | blue;
    return val;

    // printf("%d %d %d\n", red, green, blue);
}

void Camera::write_framebuffer() {
    FILE* file = fopen(image_name, "w");
    fprintf(file, "P3\n%zu %zu\n255\n", image_width, image_height);

    for (size_t row = 0; row < image_height; row++) {
        fprintf(stderr, "\rScanlines remaining: %zu    ", (image_height - row));
        fflush(stderr);

        for (size_t col = 0; col < image_width; col++) {
            auto ind = row * image_width + col;
            uint8_t red = (frame_buffer[ind] >> 16) & 0xff;
            uint8_t green = (frame_buffer[ind] >> 8) & 0xff;
            uint8_t blue = frame_buffer[ind] & 0xff;
            fprintf(file, "%d %d %d\n", red, green, blue);
        }
    }

    fclose(file);
    free(frame_buffer);
    fprintf(stderr, "\rDone                    \n");
}