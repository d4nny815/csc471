#include "hip/hip_runtime.h"
#include "camera.h"

/**
 * @brief Constructs a Camera object with specified parameters.
 * @param aspect_ratio The aspect ratio of the image (width/height).
 * @param image_width The width of the output image in pixels.
 * @param samples_per_pixel The number of samples per pixel for anti-aliasing.
 * @param child_rays The number of child rays for each primary ray.
 * @param fov_deg The field of view in degrees, smaller zooms in
 * @param position The position of the camera in world space.
 * @param look_at The point the camera is looking at.
 * @param up_vector The up direction for the camera.
*/
Camera::Camera(char* filename, float aspect_ratio, size_t image_width, size_t samples_per_pixel, 
    size_t child_rays, float vfov_deg, float defocus_angle_deg, float focus_dist,
    const point3& position, const point3& look_at, const point3& up_vector) : 
    image_name(filename), image_width(image_width), 
    image_height(static_cast<size_t>(image_width / aspect_ratio)),
    aspect_ratio(aspect_ratio), samples_per_pixel(samples_per_pixel),
    scale_per_pixel(1.0f / samples_per_pixel), child_rays(child_rays), 
    defocus_angle_deg(defocus_angle_deg), focus_dist(focus_dist),
    pos(position), look_at(look_at), up_vector(up_vector) {

    // calc focal length
    // focal_length = (position - look_at).length();

    // calc viewport dimensions based on field of view
    vfov = degrees_to_radians(vfov_deg);
    float h = tan(vfov / 2);
    viewport_height = 2 * h * focus_dist;
    viewport_width = viewport_height * aspect_ratio;

    // calc orthonormal basis vectors for the camera coordinate system
    w = unit_vector(position - look_at);
    u = unit_vector(cross(up_vector, w));
    v = cross(w, u);

    // calc viewport edge vectors
    v_u = viewport_width * u;
    v_v = viewport_height * -v;

    // calc pixel delta vectors
    pixel_du = v_u / image_width;
    pixel_dv = v_v / image_height;

    // calc upper-left corner of the viewport
    viewport_upper_left = position - (focus_dist * w) - v_u / 2 - v_v / 2;
    pixel00_loc = viewport_upper_left + 0.5f * (pixel_du + pixel_dv);
    
    // Calculate the camera defocus disk basis vectors.
    auto defocus_radius = focus_dist * tan(degrees_to_radians(defocus_angle_deg / 2));
    defocus_disk_u = u * defocus_radius;
    defocus_disk_v = v * defocus_radius;

    frame_buffer = static_cast<uint32_t*>(malloc(sizeof(uint32_t) * 
        image_height * image_width));
}

/**
 * @brief shoot ray from cam to a pixel in the viewport
 * @param col current col in the viewport
 * @param row current row in the viewport
 * @return ray originating from the camera to the sampled pixel
 */
Ray Camera::get_ray(size_t col, size_t row) {
    // add random jitter to enable anti-aliasing
    vec3 offset = sample_square(); 

    auto u = col + offset.x();
    auto v = row + offset.y();

    // get pixel location in the viewport
    vec3 pixel_sample = pixel00_loc + (u * pixel_du) + (v * pixel_dv);
    
    // dir from the cam to the sample pixel
    point3 ray_ori = (defocus_angle_deg <= 0) ? pos : defocus_disk_sample();
    vec3 ray_dir = pixel_sample - ray_ori;

    return Ray(ray_ori, ray_dir);
}

/**
 * 
 */
point3 Camera::defocus_disk_sample() const {
    vec3 p = random_in_unit_disk();
    return pos + (p.x() * defocus_disk_u) + (p.y() * defocus_disk_v);
}

__global__ void render_kernel(Color* framebuffer, int image_width, int image_height, 
    int samples_per_pixel, Camera camera, HittableList world, 
    float scale_per_pixel) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= image_width || row >= image_height) return;

    Color pixel_color = Color(0, 0, 0);

    // Monte Carlo sampling for anti-aliasing
    for (int i = 0; i < samples_per_pixel; i++) {
    Ray r = camera.get_ray(col, row);
    pixel_color += camera.ray_color(r, camera.child_rays, world);
    }

    // Average the accumulated color
    framebuffer[row * image_width + col] = pixel_color * scale_per_pixel;
}


Color Camera::ray_color(const Ray& r, const size_t depth, const HittableList& world) 
    const {
    
    Ray current_ray = r;

    const Color SKY_BLUE = Color(0.5, 0.7, 1.0);
    vec3 unit_direction = unit_vector(current_ray.dir);
    auto a = 0.5 * (unit_direction.y() + 1.0);
    return ((1.0 - a) * WHITE + a * SKY_BLUE);
}

/**
 * @brief return vector with x = [-.5, .5] and y = [-.5, .5] 
 */
inline vec3 Camera::sample_square() {
    return vec3(rand_float() - 0.5, rand_float() - 0.5, 0);
}

/**
 * @brief render the scene
 * 
 * shoot multiple rays from cam to each pixel in the viewport to the world
 * 
 * @param world hittable objects in the world 
 */
void Camera::render(const HittableList& world) {
    int num_pixels = image_width * image_height;
    Color* d_framebuffer;
    
    // Allocate memory on the GPU
    hipMalloc(&d_framebuffer, num_pixels * sizeof(Color));

    // Define CUDA block and grid dimensions
    dim3 block_size(16, 16);
    dim3 grid_size((image_width + block_size.x - 1) / block_size.x, 
                   (image_height + block_size.y - 1) / block_size.y);

    // Launch CUDA kernel
    render_kernel<<<grid_size, block_size>>>(d_framebuffer, image_width, image_height, 
                                             samples_per_pixel, *this, world, scale_per_pixel);

    // Copy framebuffer from GPU to CPU
    Color* h_framebuffer = new Color[num_pixels];
    hipMemcpy(h_framebuffer, d_framebuffer, num_pixels * sizeof(Color), hipMemcpyDeviceToHost);

    // Write framebuffer to file or screen
    write_framebuffer(h_framebuffer);

    // Cleanup
    delete[] h_framebuffer;
    hipFree(d_framebuffer);
}

/**
 * @brief write the computed color value to the output in PPM format
 * 
 * takes a color vector, applies gamma correction, clamps
 * the values to the valid range, and converts them to 8bit color values and 
 * prints them for PPM format
 * 
 * @param k The color vector containing RGB values in linear space.
 */
void Camera::write_color(Color k, size_t row, size_t col) {
    static const Interval color_int(0, .999); 

    float r = linear_to_gamma(k.x());
    float g = linear_to_gamma(k.y());
    float b = linear_to_gamma(k.z());

    uint8_t red = uint8_t(256 * color_int.clamp(r));
    uint8_t green = uint8_t(256 * color_int.clamp(g));
    uint8_t blue = uint8_t(256 * color_int.clamp(b));

    uint32_t val = (red << 16) | (green << 8) | blue;
    frame_buffer[row * image_width + col] = val;

    // printf("%d %d %d\n", red, green, blue);
}

void Camera::write_framebuffer() {
    FILE* file = fopen(image_name, "w");
    fprintf(file, "P3\n%zu %zu\n255\n", image_width, image_height);

    for (size_t row = 0; row < image_height; row++) {
        fprintf(stderr, "\rScanlines remaining: %zu    ", (image_height - row));
        fflush(stderr);

        for (size_t col = 0; col < image_width; col++) {
            auto ind = row * image_width + col;
            uint8_t red = (frame_buffer[ind] >> 16) & 0xff;
            uint8_t green = (frame_buffer[ind] >> 8) & 0xff;
            uint8_t blue = frame_buffer[ind] & 0xff;
            fprintf(file, "%d %d %d\n", red, green, blue);
        }
    }

    fclose(file);
    free(frame_buffer);
    fprintf(stderr, "\rDone                    \n");
}

/**
 * @brief convert linear color component to correct gamma-corrected color
 * 
 * the standard gamma correction assumes a gamma of 2.0, which is achieved 
 * by taking the square root of the component.
 * 
 */
inline float linear_to_gamma(float linear_comp) {
    return linear_comp > 0 ? std::sqrt(linear_comp) : 0;
}
